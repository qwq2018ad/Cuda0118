#include "hip/hip_runtime.h"
﻿
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <string.h>
#include <stdlib.h>
#include <iostream>

#include <fstream>
#include <sstream>
#include <time.h>
#include <chrono>

#define MAX_BLOCK_SIZE 1024 // 每個 block 的最大執行緒數量

// 計算 Hamming 距離的 Kernel
__global__ void computeHammingDistances(
    const unsigned long long* bitText,       // 位元串陣列
    const int* posTemp,            // 字串位置陣列
    int numberInputString,         // 總字串數量
    int parameterL,                // 漢明距離的參數 L
    unsigned long long bitReference,         // 參考字串的位元串
    unsigned long long clearWitnessBit,
    unsigned long long keepWitnessBit,
    int* distance                  // 輸出：每個字串的漢明距離
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numberInputString) return; // 超出範圍則退出

    // 計算字串的位元串
    int aux1 = numberSymbolInUll - posTemp[idx] % numberSymbolInUll;
    int aux2 = posTemp[idx] / numberSymbolInUll;
    unsigned long long bitString;

    if (aux1 >= parameterL) {
        bitString = (bitText[aux2] >> ((aux1 - parameterL) * numberBitForSymbol)) & filterUselessBit;
    }
    else {
        bitString = ((bitText[aux2] << ((parameterL - aux1) * numberBitForSymbol)) |
            (bitText[aux2 + 1] >> ((numberSymbolInUll - parameterL + aux1) * numberBitForSymbol))) &
            filterUselessBit;
    }

    // 漢明距離計算 (popcount 方法) 
    distance[idx] = __popcll(((bitString ^ bitReference) + clearWitnessBit)& keepWitnessBit);
}

__global__ void countingSort(
    const int* distance,  // 輸入：漢明距離陣列
    int* counting,        // 輸出：計數排序表
    int numberInputString,
    int parameterL
) {
    __shared__ int localCounting[128]; // 共享記憶體（根據最大 L 設置）
    int idx = threadIdx.x;

    // 初始化共享記憶體
    if (idx < parameterL + 1) localCounting[idx] = 0;
    __syncthreads();

    // 累計漢明距離的出現次數
    for (int i = threadIdx.x; i < numberInputString; i += blockDim.x) {
        atomicAdd(&localCounting[distance[i]], 1);
    }
    __syncthreads();

    // 將結果寫回全域記憶體
    if (idx < parameterL + 1) atomicAdd(&counting[idx], localCounting[idx]);
}

void parallelGroupStrings(
    const uint64_t* bitText,
    const int* posTemp,
    int numberInputString,
    int parameterL,
    uint64_t bitReference
) {
    int* d_distance, * d_counting;
    uint64_t* d_bitText;
    int* d_posTemp;

    // 記憶體分配
    hipMalloc(&d_distance, sizeof(int) * numberInputString);
    hipMalloc(&d_counting, sizeof(int) * (parameterL + 1));
    hipMalloc(&d_bitText, sizeof(uint64_t) * textSize);
    hipMalloc(&d_posTemp, sizeof(int) * numberInputString);

    hipMemcpy(d_bitText, bitText, sizeof(uint64_t) * textSize, hipMemcpyHostToDevice);
    hipMemcpy(d_posTemp, posTemp, sizeof(int) * numberInputString, hipMemcpyHostToDevice);
    hipMemset(d_counting, 0, sizeof(int) * (parameterL + 1));

    // 設定執行緒和區塊數量
    int blockSize = 256;
    int numBlocks = (numberInputString + blockSize - 1) / blockSize;

    // 啟動 CUDA 核心
    computeHammingDistances << <numBlocks, blockSize >> > (d_bitText, d_posTemp, numberInputString, parameterL, bitReference, d_distance);
    countingSort << <1, 128 >> > (d_distance, d_counting, numberInputString, parameterL);

    // 複製結果回主機
    int* counting = new int[parameterL + 1];
    hipMemcpy(counting, d_counting, sizeof(int) * (parameterL + 1), hipMemcpyDeviceToHost);

    // 清理記憶體
    hipFree(d_distance);
    hipFree(d_counting);
    hipFree(d_bitText);
    hipFree(d_posTemp);

    // 輸出計數表
    for (int i = 0; i <= parameterL; i++) {
        printf("Distance %d: %d\n", i, counting[i]);
    }
    delete[] counting;
}


/* Define the maximum size of alphabet */
#define MAX_ALPHABET_SIZE 255
/* Define boolean values */
#define true 1
#define false 0

/* Define the abbreviated names */
typedef unsigned char UC;
typedef unsigned long long ULL;

/* Define the struct for a node in the reference tree */
/* Note that this struct is for internal and leaf nodes */
typedef struct _REFERENCE_TREE_NODE
{
    int numberReference;                     /* The number of input strings */
    int reference;                           /* The reference string */
    bool order;                              /* The flag for accessing memory */
    UC children;                             /* The number of child nodes (which will be expanded)*/
    UC* distance;                            /* The child nodes' distances (with the reference string) */
    struct _REFERENCE_TREE_NODE** childNode; /* pointers for child nodes */
} ReferenceTreeNode;

/* Define the struct for an element in job stack */
typedef struct _STACK
{
    int numberInputString;                   /* The number of input strings */
    int start;                               /* The start position for the memory */
    bool order;                              /* The flag for accessing memory */
    void* pointerNode;                       /* The child node needed to be expaned */
    struct _STACK* previous;                 /* The previous element in the job stack */
    struct _STACK* next;                     /* The next element in the job stack */
} Stack;

/* Calculate the number of bits whose values are 1 in a computer word */
int ComputeHD_64(ULL i)
{
    i = i - ((i >> 1) & 0x5555555555555555UL);
    i = (i & 0x3333333333333333UL) + ((i >> 2) & 0x3333333333333333UL);
    return (int)((((i + (i >> 4)) & 0xF0F0F0F0F0F0F0FUL) * 0x101010101010101UL) >> 56);
}

/* Free the memory of the node in the reference tree */
void FreeTree(ReferenceTreeNode** node)
{
    /* If any subtree belongs to this node, the memory for the subtree will be free recurrsively. */
    if ((*node)->children > 1)
    {
        int i = 0;
        for (; i < (*node)->children - 1; i++)
            FreeTree(&((*node)->childNode[i]));
        free((*node)->childNode);
        free((*node)->distance);
    }
    /* After the recursive free, the memory of this node is freed */
    free(*node);
}

int main(int argc, char** argv)
{
    int iterations = 5;
    double totalTime1 = 0.0;
    double totalTime2 = 0.0;
    double totalTime3 = 0.0;
    for (int kk = 0; kk < iterations; kk++) {
        UC** pattern = NULL;                 /* The array for recording patterns */
        int* lengthPattern = NULL;           /* The array for recording lengths of pattern */
        int numberPattern = 0;            /* The number of patterns */
        //FILE* pointerPatternFile = NULL;     /* The FILE pointer for opening the file of patterns */
        int lmin = 0;                        /* Recording the mininum length among all patterns */
        int parameterL = 0;                  /* Parameter l */
        int parameterK = 0;                  /* Parameter k */
        int i = 0;                           /* for loop index */
        clock_t preprocessingStart;   /* Time stamp for the start of preprocecssing */
        clock_t preprocessingEnd;     /* Time stamp for the end of preprocessing */
        clock_t searchingStart;       /* Time stamp for the start of searching */
        clock_t searchingEnd;         /* Time stamp for the end of searching */
        int abb, acc;
        /* Checking the number of input arguments */
        /* When the input number is incorrect, the help message for using this program is displayed. */
        if (argc != 5)
        {
            printf("Usage: %s [text file] [pattern file] [parameter l] [parameter k]\n", argv[0]);
            printf("       [text file]: the file name of the input text string\n");
            printf("       [pattern file]: the file name of the input pattern strings\n");
            printf("       [parameter l]: the user's assigned length of prefixes\n");
            printf("       [parameter k]: the user's assigned size for determining whether a node is a leaf in reference tree\n\n");
            return 0;
        }

        /* Checking whether the opening file for patterns is successfully */
        FILE* pointerPatternFile = NULL;

        // 正確使用 fopen_s
        errno_t err1 = fopen_s(&pointerPatternFile, argv[2], "rb");
        if (err1 != 0 || pointerPatternFile == NULL) {
            /* 如果無法開啟檔案，停止程式 */
            printf("ERROR: Cannot read the pattern file (\"%s\").\n", argv[2]);
            return 0;
        }



        /* Note that the format of patterns is described as follows:
            * The first line is the total number of patterns.
            * Patterns are recorded in the rest of lines.
            * Each line (from 2nd to the last lines) contains two elements.
            * The length is recorded in the first element, and its corresponding pattern is in the second element.
            * Two elements are separated by a comma.
            * */
            /* Read the number of pattern */
        fscanf_s(pointerPatternFile, "%d\n", &numberPattern);
        /* Request memory for the first dimension of pattern array */
        pattern = (UC**)malloc(sizeof(UC*) * numberPattern); // 顯式轉型為 UC**
        if (!pattern) {
            printf("Memory allocation failed for pattern\n");
            return -1;
        }
        /* Request memory for the lengths of patterns */
        lengthPattern = (int*)malloc(sizeof(int) * numberPattern); // 顯式轉型為 int*
        if (!lengthPattern) {
            printf("Memory allocation failed for lengthPattern\n");
            return -1;
        }
        /* Read each line to obtain the length and the pattern by for loop */
        for (i = 0; i < numberPattern; i++)
        {
            /* Obtain the length of the pattern first, and then request the memory of this pattern array (the second dimension) */
            fscanf_s(pointerPatternFile, "%d,", &lengthPattern[i]);
            /* Record the minimum length among all patterns */
            if (i == 0)
                lmin = lengthPattern[i];
            else if (lmin > lengthPattern[i])
                lmin = lengthPattern[i];
            pattern[i] = (UC*)malloc(lengthPattern[i] * sizeof(UC)); // 根據 lengthPattern[i] 的值來分配內存
            if (!pattern[i]) {
                printf("Memory allocation failed for pattern[i]\n");
                return -1;
            }
            /* Read the pattern and store into pattern array */
            fread(pattern[i], 1, lengthPattern[i], pointerPatternFile);
            /* Prepare for next line in the file */
            fscanf_s(pointerPatternFile, "\n");
        }
        /* close the file of patterns */
        fclose(pointerPatternFile);

        /* obtain the parameters l and k */
        parameterL = atoi(argv[3]);
        parameterK = atoi(argv[4]);

        /* set the time stamp of the start for preprocessing */
        preprocessingStart = clock();

        //FILE* pointerTextFile = NULL;          /* The FILE pointer for opening the file of text */
        int lengthText = 0;                    /* The length of text */
        UC alphabet[MAX_ALPHABET_SIZE];        /* The alphabet of strings */
        UC ENV = sizeof(ULL) * 8;              /* The size of a computer word */
        UC sizeAlphabet = 0;                   /* The size of alphabet */
        UC inputChar = 0;                      /* A temporary character for reading text string */
        UC numberBitForSymbol = 1;             /* A Symbol need how many bits*/
        UC numberSymbolInUll = 1;              /* maximum number of characters in an ULL 64/3=21 */
        ULL filterUselessBit = 0x0ULL;         /* The mask for filtering unused bits in a ULL */
        ULL clearWitnessBit = 0x0ULL;          /* The mask for filtering the witness bits */
        ULL keepWitnessBit = 0x0ULL;           /* The mask for filtering the bits whic are not witness bits */
        int totalNumberUllForText = 1;         /* The number of ULLs that text string needs */
        int numberInputString = 0;             /* The number of input strings */
        int aux1 = 0;                          /* The temporary variable for obtaining the bit string */
        int aux2 = 0;
        ULL bitReference = 0x0ULL;             /* The bit string with respect to reference string */
        ULL bitString = 0x0ULL;                /* The bit string with respect to input string */
        ULL* bitText = NULL;                   /* The bit string with respect to text string */
        int* posTemp[2];                       /* The memory for recording whole input strings (use double space) */
        UC* distance = NULL;                   /* The Hamming distances of child nodes */
        int* counting = NULL;                  /* The counting table for counting sort */
        int sizeofCounting = 0;                /* The size of counting table */
        ReferenceTreeNode* root = NULL;        /* Root node pointer */
        ReferenceTreeNode* currentNode = NULL; /* The node pointer which is ready to be expanded */
        Stack* head = NULL;                    /* The head pointer of the stack */
        Stack* currentJob = NULL;              /* The current pointer which is ready to be processing */
        int start = 0;                         /* The start position in memory for input string */
        int end = 0;                           /* The end position in memory for input string */
        int reference = 0;                     /* The position for reference string */
        UC children = 0;                       /* The number of child nodes */
        bool order = true;                     /* The flag for accessing memory */
        bool newOrder = false;                 /* The new flag for accessing memory */
        bool flag = true;
        UC HD = 0;
        int j = 0;
        int k = 0;

        /* This program is only run in 64-bit operating system. If it is not 64-bit OS, stop this program. */
        if (ENV < 64)
        {
            printf("ERROR: Please run this program in 64-bit environment.\n");
            return 0;
        }

        /* Check whether the opening file for text is successful */
        FILE* pointerTextFile = NULL;
        errno_t err2;

        if ((err2 = fopen_s(&pointerTextFile, argv[1], "rb")) != 0)
        {
            /* If fail to open the file, free the used memory and stop. */
            printf("ERROR: Cannot read the text file (\"%s\"). Error code: %d\n", argv[1], err2);
            for (i = 0; i < numberPattern; i++)
                free(pattern[i]);
            free(pattern);
            free(lengthPattern);
            return 0;
        }

        /* Read the text string and record the alphabet of text */
        memset(alphabet, 0xff, MAX_ALPHABET_SIZE);
        while ((char)(inputChar = fgetc(pointerTextFile)) != EOF)
            alphabet[inputChar] = 0;
        lengthText = ftell(pointerTextFile);  /* Obtain the length of text string */

        for (i = 0; i != MAX_ALPHABET_SIZE; i++)
            if (!alphabet[i])
                alphabet[i] = sizeAlphabet++; /* Assign an unique number for this symbol */

        for (i = 2; i < sizeAlphabet; i *= 2)
            numberBitForSymbol++;             /* Calculate the number of bits used to record a symbol */
        numberBitForSymbol++;                 /* An extra leading witness bit is needed */

        /* Note:
            * This program is only used one computer word to implement reference tree approach.
            * If the parameter l is too large that one computer word cannot deal with it, stop.
            * */
        if (parameterL * numberBitForSymbol > ENV)
        {
            printf("ERROR: The parameter l is too large! (A unit (unsigned long long) cannot handle it.)\n");
            return 0;
        }

        /* Compute the maximum number of characters in an ULL */
        numberSymbolInUll = ENV / numberBitForSymbol;
        /* Compute the number of total input strings */
        numberInputString = lengthText - (lmin > parameterL ? lmin : parameterL) + 1;
        /* Compute the number of ULLs that text string needs */
        if (lengthText % numberSymbolInUll)
            totalNumberUllForText = lengthText / numberSymbolInUll;
        else
            totalNumberUllForText = lengthText / numberSymbolInUll + 1;

        /* If characters can exactly fill up in an ULL, no bit will be filted;
            * Otherwise, compute the mask for filtering the unused bits in the high positions.
            * */
        if (parameterL * numberBitForSymbol == ENV)
            filterUselessBit = ~0x0ULL;
        else
            filterUselessBit = (0x1ULL << (parameterL * numberBitForSymbol)) - 1;

        /* Compute the masks for filtering witness bits and keeping witness bits */
        keepWitnessBit = clearWitnessBit = 0x1ULL << (numberBitForSymbol - 1);
        for (i = 1; i != parameterL; i++)
            keepWitnessBit = (keepWitnessBit << numberBitForSymbol) | clearWitnessBit;
        clearWitnessBit = (~keepWitnessBit) & filterUselessBit;

        /* Request memory for bit string with respect to text string */
        bitText = (ULL*)malloc(sizeof(ULL) * totalNumberUllForText); // 顯式轉型為 ULL*
        if (!bitText) {
            printf("Memory allocation failed for bitText\n");
            return -1;
        }
        /* Request memory for whole input strings (double space for swaping) */
        posTemp[0] = (int*)malloc(sizeof(int) * lengthText);
        posTemp[1] = (int*)malloc(sizeof(int) * lengthText);
        if (!posTemp[0]) {
            printf("Memory allocation failed for posTemp[0]\n");
            return -1;
        }
        if (!posTemp[1]) {
            printf("Memory allocation failed for posTemp[1]\n");
            return -1;
        }
        /* Request memory for recording the distances between input string and reference string */
        distance = (UC*)malloc(numberInputString);
        if (!distance) {
            printf("Memory allocation failed for distance \n");
            return -1;
        }
        /* Request memory for the table of counting sort */
        sizeofCounting = sizeof(int) * (parameterL + 1);
        counting = (int*)malloc(sizeofCounting); // 分配內存
        if (!distance) {
            printf("Memory allocation failed for distance \n");
            return -1;
        }
        printf("\n");
        //上面那行不知道是怎樣刪掉就壞了

        /* Read text string again and transfer each character into its corresponding number */
        fseek(pointerTextFile, 0, SEEK_SET);
        for (i = 0; i != lengthText; i++)
        {
            posTemp[0][i] = i;
            bitText[i / numberSymbolInUll] = (bitText[i / numberSymbolInUll] << numberBitForSymbol) | alphabet[fgetc(pointerTextFile)];
        }

        /* left shift the valid bits to the corresponding positions */
        bitText[(lengthText - 1) / numberSymbolInUll] <<= ((numberSymbolInUll - ((lengthText - 1) % numberSymbolInUll) - 1) * numberBitForSymbol);
        /* Close the file of text */
        fclose(pointerTextFile);

        /* Request memory for root node */
        root = (ReferenceTreeNode*)malloc(sizeof(ReferenceTreeNode));
        /* Request memory for the first element of stack and then initialize it */
        head = (Stack*)malloc(sizeof(Stack));
        head->start = 0;
        head->numberInputString = numberInputString;
        head->order = false;
        head->pointerNode = (void*)root;
        head->previous = NULL;
        /* Point the first element of stack to be the element ready to be processing */
        currentJob = head;

        /* Tackle the stack until the stack is empty */
        while (currentJob != NULL)
        {
            /* Dealing with the current element */
            start = currentJob->start;
            numberInputString = currentJob->numberInputString;
            order = currentJob->order;
            currentNode = (ReferenceTreeNode*)(currentJob->pointerNode);

            /* Point to next element */
            if (currentJob->previous == NULL)
            {
                free(currentJob);
                currentJob = NULL;
            }
            else
            {
                currentJob = currentJob->previous;
                free(currentJob->next);
            }

            /* Consider the number of input strings for the current node.
                * If the number is greater than parameter k, this node is an internal node needed to be expanded.
                * Otherwise, this node is a leaf node.
                * */
            if (numberInputString > parameterK) /* Internal node */
            {
                /* Calculate the ending position in the memory */
                end = start + numberInputString;
                /* Pick the first element to be reference string */
                reference = start;

                /* Initialize the table for counting sort */
                memset(counting, 0, sizeofCounting);

                /* Compute the bit string with respect the reference string */
                aux1 = numberSymbolInUll - posTemp[order][reference] % numberSymbolInUll;
                aux2 = posTemp[order][reference] / numberSymbolInUll;
                /* If all the bits of the reference string are in an ULL of the bit string with respect to text, bit string can be obtained directly.
                    * Otherwise, these bits will be extracted from two ULL of the bit string with respect to text.
                    * */
                if (aux1 >= parameterL)
                    bitReference = (bitText[aux2] >> ((aux1 - parameterL) * numberBitForSymbol)) & filterUselessBit;
                else
                    bitReference = ((bitText[aux2] << ((parameterL - aux1) * numberBitForSymbol)) | (bitText[aux2 + 1] >> ((numberSymbolInUll - parameterL + aux1) * numberBitForSymbol))) & filterUselessBit;

                /* Compute the Hamming distances between each input string and the reference string */
                for (i = start; i != end; i++)
                {
                    /* Obtain the bit string with respect to the input string
                        * This process is similar to that of reference string as shown above */
                    aux1 = numberSymbolInUll - posTemp[order][i] % numberSymbolInUll;
                    aux2 = posTemp[order][i] / numberSymbolInUll;
                    if (aux1 >= parameterL)
                        bitString = (bitText[aux2] >> ((aux1 - parameterL) * numberBitForSymbol)) & filterUselessBit;
                    else
                        bitString = ((bitText[aux2] << ((parameterL - aux1) * numberBitForSymbol)) | (bitText[aux2 + 1] >> ((numberSymbolInUll - parameterL + aux1) * numberBitForSymbol))) & filterUselessBit;
                    /* popcount method for computing Hamming distance */
                    parallelGroupStrings(bitText,  // 位元串陣列
                        lengthText,               // 文字總長度
                        parameterL,               // 參數 L
                        numberInputString        // 總字串數量
                    );
                    //HD = ComputeHD_64(((bitString ^ bitReference) + clearWitnessBit) & keepWitnessBit);
                    distance[i] = HD;
                    counting[HD]++;
                }

                /* Comuting the number of child nodes and update the table of counting sort */
                children = 0;
                for (i = 1; i <= parameterL; i++)
                {
                    if (counting[i]) children++;
                    counting[i] += counting[i - 1];
                }

                currentNode->children = children + 1;
                /* If all input string are identical, this node becomes leaf node */
                if (!children)
                {
                    currentNode->numberReference = numberInputString;
                    currentNode->reference = start;
                    currentNode->order = order;

                    continue;
                }

                /* The flag for accessingg memory of whole input strings.
                    * If the flag is 0, the first piece memory will be used.
                    * If the flag is 1, the second piece memory will be used.
                    * This approach will only copy the memory used in the further expanding easily.
                    * */
                newOrder = (order + 1) % 2;
                for (i = end - 1; i >= start; i--)
                    posTemp[newOrder][start + (--counting[distance[i]])] = posTemp[order][i];

                /* Update this internal node.
                    * 1. Record the reference string
                    * 2. Record the start position of input string in memory
                    * 3. Record the new flag of accessing memory
                    * 4. Request memory for child nodes (include their corresponding distance)
                    * */
                currentNode->numberReference = counting[1];
                currentNode->reference = start;
                currentNode->order = newOrder;

                currentNode->distance = (UC*)malloc(children);
                currentNode->childNode = (ReferenceTreeNode**)malloc(sizeof(ReferenceTreeNode*) * children);

                /* Store the processing of expanding child nodes into stack */
                for (i = 1, j = 0; i <= parameterL; i++)
                {
                    k = (i == parameterL) ? numberInputString - counting[i] : counting[i + 1] - counting[i];
                    if (!k) continue;
                    /* Request memory for the pointer of child nodes and the corrersponding distances */
                    currentNode->distance[j] = i;
                    currentNode->childNode[j] = (ReferenceTreeNode*)malloc(sizeof(ReferenceTreeNode));

                    /* Request memory for this element of stack */
                    if (currentJob == NULL)
                    {
                        currentJob = (Stack*)malloc(sizeof(Stack));
                        currentJob->previous = NULL;
                    }
                    else
                    {
                        currentJob->next = (Stack*)malloc(sizeof(Stack));
                        currentJob->next->previous = currentJob;
                        currentJob = currentJob->next;
                    }
                    /* Sotre the relative data of this element into stack */
                    currentJob->start = start + counting[i];
                    currentJob->numberInputString = k;
                    currentJob->order = newOrder;
                    currentJob->pointerNode = (void*)(currentNode->childNode[j++]);
                }
            }
            else /* Leaf node */
            {
                /* Store the input strings, the start position in the memory, and the flag of accessing memory */
                currentNode->numberReference = numberInputString;
                currentNode->reference = start;
                currentNode->order = order;
                currentNode->children = 0;
            }
        }

        /* Prepare the memory for pattern */
        ULL filterBitForSingleSymbol = (0x1ULL << numberBitForSymbol) - 0x1ULL;
        ULL bitPattern = 0x0ULL;

        /* Set the time stamp of the end for preprocessing */
        preprocessingEnd = clock();
        /* Set the time stamp of the start for searching */
        searchingStart = clock();

        /* Consider the patterns one by one */
        for (i = 0; i != numberPattern; i++)
        {
            /* Initialize the bit string for pattern */
            bitPattern = 0x0ULL;
            /* Compute the bit string with respect to this pattern */
            for (j = 0; j != parameterL; j++)
                bitPattern = (bitPattern << numberBitForSymbol) | alphabet[pattern[i][j]];

            /* Search the reference tree started from root node */
            currentNode = root;
            while (1)
            {   /* 1. 開啟檔案
                FILE* file = fopen("output.txt", "w");
                if (file == NULL) {
                    perror("無法開啟檔案");
                    return 1;
                }*/
                /* If this is an internal node, compute the Hamming distance between l-prefix of this pattern and the reference string */
                if (currentNode->children != 0)
                {
                    /* Obtain the bit string with respect to the reference string */
                    aux1 = numberSymbolInUll - posTemp[currentNode->order][currentNode->reference] % numberSymbolInUll;
                    aux2 = posTemp[currentNode->order][currentNode->reference] / numberSymbolInUll;
                    if (aux1 >= parameterL)
                        bitReference = (bitText[aux2] >> ((aux1 - parameterL) * numberBitForSymbol)) & filterUselessBit;
                    else
                    {
                        //                    if (aux2 + 2 == totalNumberUllForText)
                        //                        bitReference = ((bitText[aux2] << ((parameterL - aux1) * numberBitForSymbol)) | (bitText[aux2 + 1] >> ((numberSymbolInLastUllForText - parameterL + aux1) * numberBitForSymbol))) & filterUselessBit;
                        //                    else
                        bitReference = ((bitText[aux2] << ((parameterL - aux1) * numberBitForSymbol)) | (bitText[aux2 + 1] >> ((numberSymbolInUll - parameterL + aux1) * numberBitForSymbol))) & filterUselessBit;
                    }
                    

                    /* Compute the Hamming distance by popcount method */
                    HD = ComputeHD_64(((bitPattern ^ bitReference) + clearWitnessBit) & keepWitnessBit);
                    if (HD) /* If the distance is not zero, the corresponding subtree will be searched */
                    {
                        start = currentNode->children - 1;
                        /* If the child node whose distance is equal to the current Hamming distance exists, continue the search.
                            * Otherwise (such child node does not exist), the search for this pattern stops.
                            * */
                        flag = false;
                        for (j = 0; j != start; j++)
                            if (currentNode->distance[j] == HD)
                            {
                                flag = true;
                                break;
                            }
                        if (flag == true)
                        {
                            currentNode = currentNode->childNode[j];
                            continue;
                        }
                        else
                            break;
                    }
                    else /* If the distance is zero, search go to the special leaf node whose strings all are identical to the l-prefix of this pattern */
                    {
                        /* For each string in this node, linear compare with the remaining suffixes of patterns one by one */
                        end = currentNode->reference + currentNode->numberReference;
                        order = currentNode->order;
                        /* If the length of pattern is equal to parameter l, exact matches are found */
                        if (parameterL == lengthPattern[i])
                        {
                            for (j = currentNode->reference; j != end; j++)
                                //fprintf(file, "%d(%d),", posTemp[order][j] + parameterL, i + 1);
                                printf("%d(%d),", posTemp[order][j] + parameterL, i + 1);
                                abb = 0;
                        }
                        else
                        {
                            for (start = currentNode->reference; start != end; start++)
                            {
                                flag = true;
                                for (j = parameterL, k = posTemp[order][start] + parameterL; j < lengthPattern[i]; j++, k++)
                                    if (alphabet[pattern[i][j]] != ((bitText[k / numberSymbolInUll] >> ((numberSymbolInUll - k % numberSymbolInUll - 1) * numberBitForSymbol)) & filterBitForSingleSymbol))
                                    {
                                        flag = false;
                                        break;
                                    }
                                if (flag == true)
                                {
                                    if (k <= lengthText)
                                        //fprintf(file, "%d(%d),", k, i + 1);
                                        printf("%d(%d),", k, i + 1);
                                        acc = 0;
                                }
                            }
                        }

                        break;
                    }
                }
                /* If this node is leaf node, compare the string in this node with pattern one by one */
                else
                {
                    end = currentNode->reference + currentNode->numberReference;
                    order = currentNode->order;
                    for (start = currentNode->reference; start != end; start++)
                    {
                        flag = true;
                        for (j = 0, k = posTemp[order][start]; j < lengthPattern[i]; j++, k++)
                            if (alphabet[pattern[i][j]] != ((bitText[k / numberSymbolInUll] >> ((numberSymbolInUll - k % numberSymbolInUll - 1) * numberBitForSymbol)) & filterBitForSingleSymbol))
                            {
                                flag = false;
                                break;
                            }
                        if (flag == true)
                        {
                            if (k <= lengthText)
                                //fprintf(file, "%d(%d),", k, i + 1);
                                printf("%d(%d),", k, i + 1);
                                acc = 0;
                        }
                    }

                    break;
                }
            }
        }

        /* 預處理(建樹)、搜尋、全部時間*/
        searchingEnd = clock();
        double preprocessing_clock = (static_cast<double>(preprocessingEnd - preprocessingStart) / CLOCKS_PER_SEC) * 1000.0;
        double searching_clock = (static_cast<double>(searchingEnd - searchingStart) / CLOCKS_PER_SEC) * 1000.0;
        double all_clock = (static_cast<double>(searchingEnd - preprocessingStart) / CLOCKS_PER_SEC) * 1000.0;

        totalTime1 += preprocessing_clock;
        totalTime2 += searching_clock;
        totalTime3 += all_clock;

        printf("%.2f, %.2f, %.2f\n", preprocessing_clock, searching_clock, all_clock);
        /* Free the memories*/
        for (i = 0; i < numberPattern; i++)
            free(pattern[i]);
        free(pattern);
        free(lengthPattern);
        free(bitText);
        free(posTemp[0]);
        free(posTemp[1]);
        free(distance);
        free(counting);
        FreeTree(&root);

    }
    double averageTime1 = totalTime1 / iterations;
    double averageTime2 = totalTime2 / iterations;
    double averageTime3 = totalTime3 / iterations;
    printf("average:\n");
    printf("%f, %f ,%f\n", averageTime1, averageTime2, averageTime3);
    /* Return normal ending to system */
    return 0;
}
